#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <set>
#include <sstream>
#include <string>
#include <fstream>
#include <iostream>
#include <cstring>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;


__global__ void GraphGenerator(int* matrix,int* dimension, int* address, int* h_graph, int V)
{
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(int i = index; i < V; i += stride)
  {
    int a = address[i];
    int j = 0;
    for (int k = 0; k < V; k++)
    {
      if (matrix[i*V + k])
      {
        h_graph[a + j] = k;
        j++;
      }
    }
  }
}

__global__ void DimensionGenerator(int* matrix, int* dimension, int* address, int V)
{
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(int i = index; i < V; i += stride)
  {
    for (int j = 0; j < V; j++)
    {
      if(matrix[i*V + j])
      {
        dimension[i]++;
      }
    }
  }
  __syncthreads();
}


//================================Utility Functions=======================================
void CountColors(int V,int length, int* color, int &minColors, int &minIndex)
{
	//int minColors = INT_MAX;
	//int minIndex;
   int *num_colors;
	num_colors = (int*) malloc(sizeof(int) * length);
	for (int i = 0; i < length; i++)
	{
		num_colors[i] = 0;
	}
   set<int> seen_colors;

   for (int i = 0; i < length; i++) {
      if (seen_colors.find(color[i]) == seen_colors.end())
      {
         seen_colors.insert(color[i]);
         num_colors[i/V]++;
      }
      if(i%V==V-1)
      {
        //cout<<num_colors[i/V]<<endl;
	if (num_colors[i/V] < minColors)
	{
		minColors = num_colors[i/V];
		minIndex = i / V;
	}
        seen_colors.clear();
        //num_colors = 0;
      }
   }
}

bool IsValidColoring(int* graph, int V, int* color)
{
   for (int i = 0; i < V; i++) {
      for (int j = 0; j < V; j++) {
         if (graph[i * V + j]) {
            if (i != j && color[i] == color[j]) {
               printf("Vertex %d and Vertex %d are connected and have the same color %d\n", i, j, color[i]);
               return false;
            }
            if (color[i] < 1) {
               printf("Vertex %d has invalid color %d\n", i, color[i]);

            }
         }
      }
   }

   return true;
}

//Load raw .co data
void getDimension(const char filename[], int* V)
{
   string line;
   ifstream infile(filename);
   if (infile.fail()) {
      printf("Failed to open %s\n", filename);
      return;
   }

   int num_rows;

   while (getline(infile, line))
   {
      istringstream iss(line);
      string s;
      iss >> s;
      if (s == "p") {
         iss >> s; // read string "edge"
         iss >> num_rows;
         *V = num_rows;
         break;
      }
   }
   infile.close();
}

void ReadColFile(const char filename[], int* graph, int V)
{
   string line;
   ifstream infile(filename);
   if (infile.fail()) {
      printf("Failed to open %s\n", filename);
      return;
   }

   while (getline(infile, line)) {
      istringstream iss(line);
      string s;
      int node1, node2;
      iss >> s;
      if (s != "e")
         continue;

      iss >> node1 >> node2;

      // Assume node numbering starts at 1
      (graph)[(node1 - 1) * V + (node2 - 1)] = 1;
      (graph)[(node2 - 1) * V + (node1 - 1)] = 1;
   }
   infile.close();
}

//print graph Matrix
void PrintMatrix(int* matrix, int M, int N) {
   for (int row=0; row<M; row++)
   {
      for(int columns=0; columns<N; columns++)
      {
         printf("%i", matrix[row * N + columns]);
      }
      printf("\n");
   }
}


// Read MatrixMarket graphs
// Assumes input nodes are numbered starting from 1
void ReadMMFile(const char filename[], bool** graph, int* V)
{
   string line;
   ifstream infile(filename);
   if (infile.fail()) {
      printf("Failed to open %s\n", filename);
      return;
   }

   // Reading comments
   while (getline(infile, line)) {
      istringstream iss(line);
      if (line.find('%') == string::npos)
         break;
   }

   // Reading metadata
   istringstream iss(line);
   int num_rows, num_cols, num_edges;
   iss >> num_rows >> num_cols >> num_edges;

   *graph = new bool[num_rows * num_rows];
   memset(*graph, 0, num_rows * num_rows * sizeof(bool));
   *V = num_rows;

   // Reading nodes
   while (getline(infile, line)) {
      istringstream iss(line);
      int node1, node2, weight;
      iss >> node1 >> node2 >> weight;

      // Assume node numbering starts at 1
      (*graph)[(node1 - 1) * num_rows + (node2 - 1)] = true;
      (*graph)[(node2 - 1) * num_rows + (node1 - 1)] = true;
   }
   infile.close();
}


//Constraints
int* const2(int numVertices)
{
  int *a;
  a = (int*) malloc(sizeof(int) * numVertices);
  for (int i = 0; i < numVertices; i++)
    a[i] = 1;
  return a;
}
//===================================Main=======================================
void GraphColoringGPU(const char filename[])
{
  int * matrix;
  int * h_graph;
  int * sequence;
  int * dimension;
  int * address;
  int * result;
  int V;



  if (string(filename).find(".col") != string::npos)
  {
    getDimension(filename, &V);
    hipError_t result = hipMallocManaged(&matrix,sizeof(int)*V*V);
    ReadColFile(filename,matrix,V);
  }
  /*
  else if (string(filename).find(".mm") != string::npos)
     ReadMMFile(filename, matrix, V);*/


  hipMallocManaged(&sequence, sizeof(int) * V );
  hipMallocManaged(&dimension,sizeof(int)*V);
  hipMallocManaged(&address,sizeof(int)*V);
  hipMallocManaged(&result, sizeof(int) *V);


  DimensionGenerator<<<256,1024>>>(matrix,dimension,address,V);
  hipDeviceSynchronize();
  thrust::exclusive_scan(thrust::host,dimension,&dimension[V],address);
  hipMallocManaged(&h_graph,sizeof(int)* (dimension[V-1]+address[V-1]));

  GraphGenerator<<<256,1024>>>(matrix,dimension,address,h_graph,V);
  hipDeviceSynchronize();


  hipFree(h_graph);
  hipFree(dimension);
  hipFree(sequence);
  hipFree(address);
  hipFree(matrix);
  hipFree(result);

}

int main(int argc, char const *argv[]) {

  GraphColoringGPU(argv[1]);

  return 0;
}
