#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <set>
#include <sstream>
#include <string>
#include <fstream>
#include <iostream>
#include <cstring>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

//
// __global__ void GraphGenerator(int* matrix,int* dimension, int* address, int* h_graph, int V)
// {
//   int index = threadIdx.x + blockDim.x * blockIdx.x;
//   int stride = blockDim.x * gridDim.x;
//   for(int i = index; i < V; i += stride)
//   {
//     int a = address[i];
//     int j = 0;
//     for (int k = 0; k < V; k++)
//     {
//       if (matrix[i*V + k])
//       {
//         h_graph[a + j] = k;
//         j++;
//       }
//     }
//   }
// }
//
// __global__ void DimensionGenerator(int* matrix, int* dimension, int* address, int V)
// {
//   int index = threadIdx.x + blockDim.x * blockIdx.x;
//   int stride = blockDim.x * gridDim.x;
//   for(int i = index; i < V; i += stride)
//   {
//     for (int j = 0; j < V; j++)
//     {
//       if(matrix[i*V + j])
//       {
//         dimension[i]++;
//       }
//     }
//   }
//   __syncthreads();
// }
//
//
// //================================Utility Functions=======================================
// void CountColors(int V,int length, int* color, int &minColors, int &minIndex)
// {
// 	//int minColors = INT_MAX;
// 	//int minIndex;
//    int *num_colors;
// 	num_colors = (int*) malloc(sizeof(int) * length);
// 	for (int i = 0; i < length; i++)
// 	{
// 		num_colors[i] = 0;
// 	}
//    set<int> seen_colors;
//
//    for (int i = 0; i < length; i++) {
//       if (seen_colors.find(color[i]) == seen_colors.end())
//       {
//          seen_colors.insert(color[i]);
//          num_colors[i/V]++;
//       }
//       if(i%V==V-1)
//       {
//         //cout<<num_colors[i/V]<<endl;
// 	if (num_colors[i/V] < minColors)
// 	{
// 		minColors = num_colors[i/V];
// 		minIndex = i / V;
// 	}
//         seen_colors.clear();
//         //num_colors = 0;
//       }
//    }
// }
//
// bool IsValidColoring(int* graph, int V, int* color)
// {
//    for (int i = 0; i < V; i++) {
//       for (int j = 0; j < V; j++) {
//          if (graph[i * V + j]) {
//             if (i != j && color[i] == color[j]) {
//                printf("Vertex %d and Vertex %d are connected and have the same color %d\n", i, j, color[i]);
//                return false;
//             }
//             if (color[i] < 1) {
//                printf("Vertex %d has invalid color %d\n", i, color[i]);
//
//             }
//          }
//       }
//    }
//
//    return true;
// }
//
// //Load raw .co data
// void getDimension(const char filename[], int* V)
// {
//    string line;
//    ifstream infile(filename);
//    if (infile.fail()) {
//       printf("Failed to open %s\n", filename);
//       return;
//    }
//
//    int num_rows;
//
//    while (getline(infile, line))
//    {
//       istringstream iss(line);
//       string s;
//       iss >> s;
//       if (s == "p") {
//          iss >> s; // read string "edge"
//          iss >> num_rows;
//          *V = num_rows;
//          break;
//       }
//    }
//    infile.close();
// }


// //print graph Matrix
// void PrintMatrix(int* matrix, int M, int N) {
//    for (int row=0; row<M; row++)
//    {
//       for(int columns=0; columns<N; columns++)
//       {
//          printf("%i", matrix[row * N + columns]);
//       }
//       printf("\n");
//    }
// }


// Read MatrixMarket graphs
// // Assumes input nodes are numbered starting from 1
// void ReadMMFile(const char filename[], bool** graph, int* V)
// {
//    string line;
//    ifstream infile(filename);
//    if (infile.fail()) {
//       printf("Failed to open %s\n", filename);
//       return;
//    }
//
//    // Reading comments
//    while (getline(infile, line)) {
//       istringstream iss(line);
//       if (line.find('%') == string::npos)
//          break;
//    }
//
//    // Reading metadata
//    istringstream iss(line);
//    int num_rows, num_cols, num_edges;
//    iss >> num_rows >> num_cols >> num_edges;
//
//    *graph = new bool[num_rows * num_rows];
//    memset(*graph, 0, num_rows * num_rows * sizeof(bool));
//    *V = num_rows;
//
//    // Reading nodes
//    while (getline(infile, line)) {
//       istringstream iss(line);
//       int node1, node2, weight;
//       iss >> node1 >> node2 >> weight;
//
//       // Assume node numbering starts at 1
//       (*graph)[(node1 - 1) * num_rows + (node2 - 1)] = true;
//       (*graph)[(node2 - 1) * num_rows + (node1 - 1)] = true;
//    }
//    infile.close();
// }


//Constraints=======================================================================================
//
// int* const2(int numVertices) //This requires transpose.
// {
//   int *toRet;
//   toRet = (int*) malloc(sizeof(int) * numVertices);
//   for (int i = 0; i < numVertices; i++)
//     toRet[i] = 1;
//   return toRet;
// }

// int* allConsts(int * matrix, int numVertices)
// {
//   int *toRet;
//   int numEdges = 0;
//   for (int i = 0; i < numVertices * numVertices; i++)
//   {
//     if(matrix[i])
//       numEdges++;
//   }
//
//   hipMallocManaged(*toRet, sizeof((int) * ((numEdges+5)*numVertices)* (4*numVertices+4));
//   // toRet = (int*) malloc(sizeof(int) * ((numEdges+5)*numVertices)* (4*numVertices+4));//Y dimension is 6*numVertices and X dimension is 2*numVertices;
//
//   int row;
//   int col;
//   //Constraint 1
//   for (int i = 0; i < numEdges * numVertices; i++)
//   {
//
//   }
//
//   //Constraint 2
//   int startConstraint2 = i % (4*numVertices + 4); //The row at which const 2 matches.
//   for (; i < numVertices; i++)
//   {
//     row = i % (4*numVertices + 4);
//     col = i / (4*numVertices + 4);
//     toRet[]
//     for (int j = 0; j < 4; j++)
//     {
//       toRet[]
//     }
//   }
// }

// int* const1(int * matrix, int numVertices)
// {
//   //Find # of edges.
//   int * toRet;
//   int numEdges = 0;
//   for (int i = 0; i < numVertices * numVertices; i++)
//   {
//     if(matrix[i])
//       numEdges++;
//   }
//   toRet = (int*) malloc(sizeof(int) * (numVertices+numEdges));
//
//   for (int i = 0; i < numVertices+numEdges; i++)
//   {
//     toRet[i] = 0;
//   }
//   //Populate the matrix to return.
//   numEdges = 0;
//   int row;
//   int col;
//   for (int i = 0; i < numVertices * numVertices; i++)
//   {
//
//     if (matrix[i])
//     {
//       row = i % numVertices;
//       col = i / numVertices;
//       toRet[numEdges* numVertices + row] = 1;
//       toRet[numEdges* numVertices + col] = 1;
//       numEdges++;
//
//     }
//
//   }
//   return toRet;
// }
void readEdgesPosition(const char filename[], int* edgeList)
{
   string line;
   ifstream infile(filename);
   if (infile.fail()) {
      printf("Failed to open %s\n", filename);
      return;
   }

   int i=0;
   while (getline(infile, line)) {
      istringstream iss(line);
      string s;
      int node1, node2;
      iss >> s;
      if (s != "e")
         continue;

      iss >> node1 >> node2;

      edgeList[2*i] = node1;
      edgeList[2*i+1] = node2;
      i++;

   }
   infile.close();
}

void getInfo(const char filename[], int* numEdges, int* numVertices)
{
   string line;
   ifstream infile(filename);
   if (infile.fail()) {
      printf("Failed to open %s\n", filename);
      return;
   }

   while (getline(infile, line)) {
      istringstream iss(line);
      string s,node1;
      iss >> s;
      if (s != "p")
         continue;

      iss >> node1 >> *numVertices >> *numEdges;

      // Assume node numbering starts at 1
   }
   infile.close();
}



__global__ void constraint1 (double *simplexTable, int* edgeList, int numColors, int numEdges, int numVertices)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(int i = index; i < numEdges; i+=stride)
  {
    for(int j = 0; j < numColors; j++)
    {
      simplexTable[(numColors*i+j)*(numVertices*numColors+numColors)+(edgeList[2*i]-1) * numColors+j] = 1.0;
      simplexTable[(numColors*i+j)*(numVertices*numColors+numColors)+(edgeList[2*i+1]-1) * numColors+j] = 1.0;
    }
  }
}

__global__ void constraint2 (double *simplexTable, int numVertices, int numColors)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = index; i < numVertices; i += stride)
  {
    for (int j = 0; j < numColors; j++)
    {
      simplexTable[i*(numVertices*numColors+numColors)+numColors*i+j] = 1.0;
    }
  }
}

__global__ void constraint3 (double *simplexTable, int numVertices, int numColors)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = index; i < numVertices*numColors; i += stride)
  {
    simplexTable[i*(numVertices*numColors+numColors) + i] = 1.0;
    simplexTable[i*(numVertices*numColors+numColors) + numColors * numVertices + (i%numColors)] = -1.0;
  }
}

//===================================Main=======================================
void constraintGenerator(const char filename[])
{
  //hipMallocManaged(&simplexTable, sizeof((int) * ((numEdges+5)*numVertices)* (4*numVertices+4))
  int numColors = 2;
  int numEdges;
  int numVertices;
  if (string(filename).find(".col") != string::npos)
  {
    getInfo(filename, &numEdges, &numVertices);
  }

  int* edgeList;
  hipMallocManaged(&edgeList,sizeof(int)*2*numEdges);
  readEdgesPosition(filename,edgeList);

  int * simp;
  hipMallocManaged(&simp, sizeof(int) *numEdges*2*10);
  for (int i = 0; i < 16*20; i++)
  {
    simp[i] = 0;
  }
   double* simplexTable;
   hipMallocManaged(&simplexTable, (numColors*numVertices + numVertices) * (numEdges*numColors + numVertices + numColors*numVertices));
  constraint1<<<1,1>>>(simplexTable, edgeList,numColors,numEdges,numVertices);
  constraint2<<<1,1>>>(simplexTable + ((numColors*numVertices + numColors)*(numEdges*numColors)), numVertices, numColors);
  constraint3<<<1,1>>>(simplexTable + ((numColors*numVertices + numColors)*(numEdges*numColors + numVertices)) , numVertices, numColors);
  hipDeviceSynchronize();
  for (int i = 0; i < (numColors*numVertices + numColors)*(numEdges*numColors + numVertices + numColors*numVertices); i++)
  {
    cout << simplexTable[i] << " ";
    if (i%10==9) cout << endl;
  }
  cout<<endl;
  hipFree(simp);
  hipFree(simplexTable);

}

int main(int argc, char const *argv[]) {


  constraintGenerator(argv[1]);
  return 0;
}
