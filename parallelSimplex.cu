#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
using namespace std;

__device__ int policy(double currentJob, double * leftChild, double * rightChild)
{
  unsigned long long seed = threadIdx.x;
  unsigned long long sequence = threadIdx.x;
  unsigned long long offset = 0;
  hiprandState_t state;
  hiprand_init(seed,sequence,offset,&state);
  if (currentJob == 1)
  {
    *leftChild = 2.;
    *rightChild = 3.;
  }
  else if (currentJob == 2)
  {
    *leftChild = 4.;
    *rightChild = 5.;
  }
  else if (currentJob == 3)
  {
    *leftChild = 6.;
    *rightChild = 7.;
  }
  int randomInt = hiprand(&state) % 10;
  // if ((randomInt) <= 9)
  //   return 0;
  // else
  if (currentJob == 1 || currentJob == 2 || currentJob == 3)
    return 1;
  else
    return 0;
}


__global__ void branchAndBound(int numblocks,int upperBound, int* doneCPU, int* lengthCPU, int* otherLengthCPU,
  double* tableListCPU, double* othertableListCPU)
{
  __shared__ int* length;
  __shared__ int* otherLength;
  __shared__ int* done;
  __shared__ double* tableList;
  __shared__ double* othertableList;
  __shared__ double currentJob;
  __shared__ int action;
  __shared__ bool localDone;
  __shared__ int index;
  __shared__ int stride;

  index = blockIdx.x;
  stride = gridDim.x;

if (threadIdx.x == 0)
{

  length = lengthCPU;
  otherLength = otherLengthCPU;
  done = doneCPU;
  tableList = tableListCPU;
  othertableList = othertableListCPU;
  printf("Thread 0 done copying.\n");
}
  // while(!(((*done2 == numblocks) && (*length1 == 0)) || ((*done1 == numblocks) && (*length2 == 0))))
  // {//if there are jobs in either array, we will keep the kernel alive.
  //
  //   localDone = false;
  //   /*The following if else statements are used to determined which array to process */
  //   if (*done1 == numblocks){
  //     done = done2;
  //     tableList = tableList2;
  //     othertableList = tableList1;
  //     length = length2;
  //     otherLength = length1;
  //   }
  //   else{
  //     done = done1;
  //     tableList = tableList1;
  //     othertableList = tableList2;
  //     length = length1;
  //     otherLength = length2;
  //   }
  //   *otherLength = 0;
  //   *done1 = 0;
  //   *done2 = 0;

    while (*done != numblocks)
    {//if not everyone is done with the current array.
      if(localDone == false)
      {//if I am not done with the current array
        for (int i = index; i < *length; i+= stride)
        {//fetch all jobs I am suppose to process
          if (threadIdx.x == 0){currentJob = tableList[i];}
          __syncthreads();

          double * leftChild, * rightChild;
          leftChild = (double*) malloc(sizeof(double));
          rightChild = (double*) malloc(sizeof (double));
          action = policy(currentJob, leftChild, rightChild);

          if (threadIdx.x == 0)
          {
            if(action == 0) // BOUND
              continue;
            else // BRANCH
            {
              atomicAdd(otherLength, 2);
              othertableList[(*otherLength) - 2] = *leftChild;
              othertableList[(*otherLength) - 1] = *rightChild;
            }
          }
          __syncthreads();
        }//after for loop, I will run out of job, I will wait for other to be done
        if (threadIdx.x == 0){atomicAdd(done, 1);localDone = true; printf("atomic add performed on done %d\n", *done);}
        __syncthreads();
      }
    }
  //}
}


int main ()
{
  double *tableList1, * tableList2;
  hipMallocManaged(&tableList1, sizeof(double) * 10);
  hipMallocManaged(&tableList2, sizeof(double) * 10);
  *tableList1 = 1.0;
  int * done1;
  hipMallocManaged(&done1, sizeof(int));
  *done1 = 0;
  int* done2;
  hipMallocManaged(&done2, sizeof(int));
  *done2 = 1;
  int * length1;
  hipMallocManaged(&length1, sizeof(int));
  *length1 = 1;
  int * length2;
  hipMallocManaged(&length2, sizeof(int));
  *length2 = 0;
  int numblocks = 1;

  //CPU Logic
  int *done;
  hipMallocManaged(&done, sizeof(int));
  double *tableList;
  hipMallocManaged(&tableList, sizeof(double) * 10);
  double *othertableList;
  hipMallocManaged(&othertableList, sizeof(double) * 10);
  int *length;
  hipMallocManaged(&length, sizeof(int));
  int *otherLength;
  hipMallocManaged(&otherLength, sizeof(int));

  while(!(((*done2 == numblocks) && (*length1 == 0)) || ((*done1 == numblocks) && (*length2 == 0))))
  {
    if (*done1 == numblocks){
      done = done2;
      tableList = tableList2;
      othertableList = tableList1;
      length = length2;
      otherLength = length1;
    }
    else{
      done = done1;
      tableList = tableList1;
      othertableList = tableList2;
      length = length1;
      otherLength = length2;
    }
    *otherLength = 0;
    *done1 = 0;
    *done2 = 0;
     printf("Kernel called\n");
    branchAndBound<<<numblocks, 1>>> (numblocks,10, done, length, otherLength, tableList, othertableList);
    hipDeviceSynchronize();
  }
  hipDeviceSynchronize();
  // branchAndBound<<<numblocks, 1>>> (numblocks,10, done1, done2, length1, length2, tableList1, tableList2);
  // hipDeviceSynchronize();
  for (int i = 0; i < 10; i++)
  {
    cout << tableList1[i] << " ";
  }
    cout << endl;

  for (int i = 0; i < 10; i++)
    cout << tableList2[i] << " ";
    cout << endl;

  hipFree(done1);
  hipFree(done2);
  hipFree(length1);
  hipFree(length2);
  hipFree(tableList1);
  hipFree(tableList2);
  hipFree(done);
  hipFree(tableList);
  hipFree(othertableList);
  hipFree(length);
  hipFree(otherLength);
  return 0;
}
