#include "hip/hip_runtime.h"
#define TOTAL_BLOCKS 256;
__global__ void branchAndBound(int upperBound,int* done1, int* done2,
int* length1, int* length2, double** TableauList1, double** TableauList2)
{
  __shared__ double* SimplexTableau;
  int* length;
  __shared__ int* done;
  __shared__ double** tablelist;
  while(*done1 != TOTAL_BLOCKS || *done2 != TOTAL_BLOCKS)//when there are still jobs
  {
    /*
      FIGURE IT OUT WHICH ARRAY TO USE HERE!!!!
    */
    if(threadIdx.x == 0){atomicAdd(*done,-1);} //signify this block is busy
    for(i = blockIdx.x; i < length; i+=gridDim.x)
    {
      if(threadIdx.x == 0)
      {
        SimplexTableau = TableauList[i];
      }
    }
    if(threadIdx.x == 0){atomicAdd(*done,1);}
  }
}
