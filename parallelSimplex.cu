#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;

__global__ void ratioCalculator(double** SimplexTableau, double* theta,
                                                double* Columnk, int k)
{
  int index = blockDim.x*blockIdx.x + threadIdx.x;
  double w = SimplexTableau[index][k];
  Columnk[index] = w;
  theta[index] = SimplexTableau[index][1]/w;
}

__global__ void normalizePivotRow( double** SimplexTableau, double* k, int k, int r)
{
  int index = blockDim.x*blockIdx.x + threadIdx.x;
  __shared__ double w;
  if(threadIdx.x == 0){w = Columnk[r];}
  __syncthreads();
  SimplexTableau[r][index] = SimplexTableau[r][index]/w;
}

__global__ void updateAllRows(double** SimplexTableau, double* Columnk, int k, int r)
{
  int xIndex = blockDim.x*blockIdx.x + threadIdx.x;
  int yIndex = blockDim.y*blockIdx.y + threadIdx.y;
  __shared__ double w[16];
  if(threadIdx.y == 0 && threadIdx.x <16)
  {
    w[threadIdx.x] = Columnk[blockIdx.y*blockDim.y+threadIdx.x];
  }
  __syncthreads();
  if(yIndex == r) return;
  SimplexTableau[yIndex][xIndex] = SimplexTableau[yIndex][xIndex] - w[threadIdx.y]
                                  *SimplexTableau[r][xIndex];
}

__global__ void updateSimplexTableau(double** SimplexTableau, double* Columnk, int k, int r)
{
  int index = blockDim.x*blockIdx.x + threadIdx.x;
  __shared__ double w;
  if(threadIdx.x == 0){w = Columnk[r]}
  __syncthreads();
  SimplexTableau[index][k] = -Columnk[index]/w;
  if(index == r){SimplexTableau[index][k]=1/w;}
}
