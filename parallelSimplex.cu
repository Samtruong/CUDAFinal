
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define TOTAL_BLOCKS 256;


__device__ int policy(double *currentJob, double * leftChild, double * rightChild)
{
  unsigned long long seed = threadIdx.x;
  unsigned long long sequence = threadIdx.x;
  unsigned long long offset = 0;
  hiprandState_t state;
  hiprand_init(seed,sequence,offset,&state);
  *leftChild = *currentJob + 1;
  *rightChild = *currentJob + 2;
  int randomInt = hiprand(&state) % 10;
  // if ((randomInt) <= 9)
  //   return 0;
  // else
     return 1;
}

__global__ void branchAndBound(int upperBound, int* done1, int* done2,
int* length1, int* length2, double** TableauList1, double** TableauList2)
{
  //__shared__ double* SimplexTableau;
  __shared__ int* length; //why pointer?
  __shared__ int* otherLength;
  __shared__ int* done;
  __shared__ double** tablelist;
  __shared__ double** otherTablelist;
  __shared__ double* currentJob;
  __shared__ int action;
  __shared__ bool localDone;
//Assign shared memory to passed in values.


  //while (*done2 == totalBlocks) //&& (length1 == 0))
   while(!(((*done2 == 256) && (length1 == 0)) || ((*done1 == 256) && (length2 == 0))))//when there are still jobs
  {
    if (*done1 == 256)
    {
      done = done2;
      tablelist = TableauList2;
      otherTablelist = TableauList1;
      length = length2;
      otherLength = length1;
      *done1 = 0;
    }

    else
    {
      done = done1;
      tablelist = TableauList1;
      otherTablelist = TableauList2;
      length = length1;
      otherLength = length2;
      *done2 = 0;
    }

    while (*done != 256)
    {
      if(!localDone)
      {
        int index = blockIdx.x;
        int stride = gridDim.x;
        for (int i = index; i < *length; i+= stride)
        {
          if (threadIdx.x == 0) //Load a job for current block.
          {
            currentJob = tablelist[i];
          }
          __syncthreads();
          double * leftChild, * rightChild;
          action = policy(currentJob, leftChild, rightChild);


          if (threadIdx.x == 0)
          {
            if(action == 0) // STOP
              continue;
            else // BRANCH
            {
              atomicAdd(otherLength, 2);
              *otherTablelist[*otherLength - 2] = *leftChild;
              *otherTablelist[*otherLength - 1] = *rightChild;
            }
          }
        }
        if (threadIdx.x == 0)
        {
          atomicAdd(done, 1);
          localDone = 1;
        }
        __syncthreads();
      }//do operation
      *length = 0;
    }
  }
}


int main ()
{
  double **TableauList1, ** TableauList2;
  int * done1 = new int(0);
  int* done2 = new int(1);
  int * length1 = new int(1);
  int * length2 = new int(0);
  branchAndBound<<<256, 1>>> (10, done1, done2, length1, length2, TableauList1, TableauList2);

  return 0;
}
