#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

#define TOTAL_BLOCKS 256;

using namespace std;

__device__ int policy(double currentJob, double * leftChild, double * rightChild)
{
  unsigned long long seed = threadIdx.x;
  unsigned long long sequence = threadIdx.x;
  unsigned long long offset = 0;
  hiprandState_t state;
  hiprand_init(seed,sequence,offset,&state);
  if (currentJob == 1)
  {
    *leftChild = 2.;
    *rightChild = 3.;
  }
  else if (currentJob == 2)
  {
    *leftChild = 4.;
    *rightChild = 5.;
  }
  else if (currentJob == 3)
  {
    *leftChild = 6.;
    *rightChild = 7.;
  }
  int randomInt = hiprand(&state) % 10;
  // if ((randomInt) <= 9)
  //   return 0;
  // else
  if (currentJob == 1 || currentJob == 2 || currentJob == 3)
    return 1;
  else
    return 0;
}

__global__ void branchAndBound(int upperBound, int* done1, int* done2,
int* length1, int* length2, double* TableauList1, double* TableauList2)
{
  // printf("hello1\n");
  //__shared__ double* SimplexTableau;
  __shared__ int* length; //why pointer?
  __shared__ int* otherLength;
  __shared__ int* done;
  __shared__ double* tablelist;
  __shared__ double* otherTablelist;
  __shared__ double currentJob;
  __shared__ int action;
  __shared__ bool localDone;
  // printf("hello2\n");
//Assign shared memory to passed in values.

int numblocks = 2;
  // printf("done2 %d, length1 %d, done1 %d, length2 %d\n", *done2, *length1, *done1, *length2);
   while(!(((*done2 == numblocks) && (*length1 == 0)) || ((*done1 == numblocks) && (*length2 == 0))))//when there are still jobs
  {
     // printf("About to enter while loop\n");
     localDone = 0;
    if (*done1 == numblocks) //start with 2
    {
      done = done2;
      tablelist = TableauList2;
      otherTablelist = TableauList1;
      length = length2;
      otherLength = length1;
      *done1 = 0;
    }

    else //start with 1
    {
      done = done1;
      tablelist = TableauList1;
      otherTablelist = TableauList2;
      length = length1;
      otherLength = length2;
      *done2 = 0;
    }
    while (*done != numblocks)
    {
      if(!localDone)
      {
        // printf("inside if\n");
        int index = blockIdx.x;
        int stride = gridDim.x;
        for (int i = index; i < *length; i+= stride)
        {
          if (threadIdx.x == 0) //Load a job for current block.
          {
            currentJob = tablelist[i];
          }
          __syncthreads();
          double * leftChild, * rightChild;
          leftChild = (double*) malloc(sizeof(double));
          rightChild = (double*) malloc(sizeof (double));
          action = policy(currentJob, leftChild, rightChild);
          // printf("otherLength %d\n", *otherLength);
          // printf("current job %f\n leftChild %f\n rightChild %f\n", currentJob, *leftChild, *rightChild);
          // printf("otherLength %d\n", *otherLength);
          if (threadIdx.x == 0)
          {
            if(action == 0) // BOUND
              continue;
            else // BRANCH
            {
              atomicAdd(otherLength, 2);
              // printf("otherLength %d\n", *otherLength);
              otherTablelist[(*otherLength) - 2] = *leftChild;
              otherTablelist[(*otherLength) - 1] = *rightChild;
              // printf("leftChild %f\n", *leftChild);
              // printf("rightChild %f\n", *rightChild);
            }
          }
        }
        if (threadIdx.x == 0)
        {
          atomicAdd(done, 1);
          localDone = 1;
        }
        __syncthreads();
      }//do operation
      *length = 0;
    }
  }
}


int main ()
{
  double *TableauList1, * TableauList2;
  hipMallocManaged(&TableauList1, sizeof(double) * 10);
  hipMallocManaged(&TableauList2, sizeof(double) * 10);
  *TableauList1 = 1.0;
  int * done1;
  hipMallocManaged(&done1, sizeof(int));
  *done1 = 0;
  int* done2;
  hipMallocManaged(&done2, sizeof(int));
  *done2 = 1;
  int * length1;
  hipMallocManaged(&length1, sizeof(int));
  *length1 = 1;
  int * length2;
  hipMallocManaged(&length2, sizeof(int));
  *length2 = 0;
  branchAndBound<<<2, 1>>> (10, done1, done2, length1, length2, TableauList1, TableauList2);

  hipDeviceSynchronize();
  for (int i = 0; i < 10; i++)
  {
    cout << TableauList1[i] << " ";
  }
    cout << endl;

  for (int i = 0; i < 10; i++)
    cout << TableauList2[i] << " ";
    cout << endl;

  hipFree(done1);
  hipFree(done2);
  hipFree(length1);
  hipFree(length2);
  hipFree(TableauList1);
  hipFree(TableauList2);
  return 0;
}
