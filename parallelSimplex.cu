#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
using namespace std;

__device__ int policy(double currentJob, double * leftChild, double * rightChild)
{
  unsigned long long seed = threadIdx.x;
  unsigned long long sequence = threadIdx.x;
  unsigned long long offset = 0;
  hiprandState_t state;
  hiprand_init(seed,sequence,offset,&state);
  if (currentJob == 1)
  {
    *leftChild = 2.;
    *rightChild = 3.;
  }
  else if (currentJob == 2)
  {
    *leftChild = 4.;
    *rightChild = 5.;
  }
  else if (currentJob == 3)
  {
    *leftChild = 6.;
    *rightChild = 7.;
  }
  int randomInt = hiprand(&state) % 10;
  // if ((randomInt) <= 9)
  //   return 0;
  // else
  if (currentJob == 1 || currentJob == 2 || currentJob == 3)
    return 1;
  else
    return 0;
}

__device__ void smallSimplexSolver(double* job, int width, int height,
   double* pivotColumn, double* pivotRow, double* ratioColumn)
{
  __shared__ double smallestObj;
  __shared__ int pivotColIndx;
  __shared__ int pivotRowIndx;
  smallestObj = 10^20;
  int index = threadIdx.x;
  int stride = blockDim.x;
  // while(true)
  // {
    //  STEP 1
    if(threadIdx.x == 0)
    {
      for(int i = 0; i < width; i++)
      {
        if(smallestObj > job[(height-1)*width+i])
        {
          smallestObj = job[(height-1)*width+i];
          pivotColIndx = i;
        }
      }
    }
    __syncthreads();
    // if(smallestObj >= 0){break;}
    printf("Smallest collumn is %d with value %f \n",pivotColIndx,smallestObj);

    for(int i = index; i < height; i+=stride)
    {
        pivotColumn[i] = job[pivotColIndx + i*width];
        ratioColumn[i] = job[(i+1)*width-1] / pivotColumn[i];
    }

  //  STEP 2
    smallestObj = 10^20;
    if(threadIdx.x == 0)
    {
      for(int i = 0; i < height-1; i++)
      {
        if(smallestObj > ratioColumn[i])
        {
          smallestObj = ratioColumn[i];
          pivotRowIndx = i;
        }
      }
    }
    __syncthreads();
    // printf("Smallest row is %d with value %f \n",pivotRowIndx,smallestObj);
    for(int i = index; i < width; i+=stride)
    {
      job[width*pivotRowIndx + i] = job[width*pivotRowIndx + i]/pivotColumn[pivotRowIndx];
      pivotRow[i] = job[width*pivotRowIndx + i];
    }
  //  STEP 3
    for (int i = index; i < height; i+= stride)
    {
      if(i == pivotRowIndx){continue;}
      for (int j = 0; j < width; j++)
      {
        job[width * i + j] = job[width * i + j] - pivotColumn[i]* pivotRow[j];
      }
    }
  //}
}


__global__ void branchAndBound(int numblocks, int* doneCPU, int* lengthCPU, int* otherLengthCPU,
  double* tableListCPU, double* othertableListCPU, int curWidth, int curHeight)
{
  __shared__ int* length;
  __shared__ double* tableList;
  __shared__ double* othertableList;
  __shared__ double* currentJob;
  __shared__ int action;
  __shared__ bool localDone;
  __shared__ int index;
  __shared__ int stride;
  __shared__ double* original;

  //simplex variables:

  index = blockIdx.x;
  stride = gridDim.x;

  if(threadIdx.x == 0)
  {
    length = lengthCPU;
    tableList = tableListCPU;
    othertableList = othertableListCPU;
  }
  __syncthreads();
  localDone = false;
  while (*doneCPU != numblocks)
  {//if not everyone is done with the current array.
    if(localDone == false)
    {//if I am not done with the current array
      for (int i = index; i < *length; i+= stride)
      {//fetch all jobs I am suppose to process
        __shared__ double* pivotRow;
        __shared__ double* pivotCol;
        __shared__ double* ratioCol;
        // c
        if(threadIdx.x == 0)
        {
          currentJob = &tableList[i*curWidth*curHeight];
          pivotCol = (double*) malloc(sizeof(double) * curHeight);
          pivotRow = (double*) malloc(sizeof(double) * curWidth);
          ratioCol = (double*) malloc(sizeof(double) * curHeight);
        }
        __syncthreads();
        original = (double*) malloc(sizeof(double) * curWidth * curHeight);
        for(int i = threadIdx.x; i < curWidth * curHeight; i+=stride)
        {
          original[i] = currentJob[i];
        }
        smallSimplexSolver(currentJob,curWidth,curHeight,pivotCol,pivotRow,ratioCol);
        for(int i = 0; i <curWidth*curHeight; i++)
        {
          printf("%f | ",(currentJob[i]));
          if(i%curWidth == curWidth-1){printf("\n");}
        }

        double  breakpoint;
        int leavingVariable;
        double* leftChild;
        double* rightChild;
        if(threadIdx.x == 0)
        {
          free(pivotCol);
          free(pivotRow);
          free(ratioCol);
          if(currentJob[curWidth*curHeight - 1] > 68.3)
          {
            action = true;
            for(int i = 0; i < curHeight - 1; i++)
            {
              if(currentJob[i + i*curWidth] == 1.)
              {
                breakpoint = currentJob[i*curHeight + curWidth - 1];
                leavingVariable = i;
                break;
              }
            }
          }
          else{action = false;}
        }
        __syncthreads();
        if(!action){continue;} //BOUND
        else //BRANCH
        {
          leftChild = (double*) malloc(sizeof(double) * (curHeight + 1) * (curWidth + 1));
          rightChild = (double*) malloc(sizeof(double) * (curHeight + 1) * (curWidth + 1));
          for(int i = threadIdx.x; i <(curHeight + 1) * (curWidth + 1); i+= blockDim.x)
          {
            leftChild[i] = 0.;
            rightChild[i] = 0.;
          }
          for(int i = threadIdx.x; i <curHeight; i+= blockDim.x)
          {
            for(int j = 0; j < curWidth-1; j++)
            {
              leftChild[i*(curWidth+1) + j] = original[i*curWidth + j];
              rightChild[i*(curWidth+1) + j] = original[i*curWidth + j];
            }
          }
          for(int i = threadIdx.x; i < curHeight; i+=blockDim.x)
          {
            leftChild[(curWidth) + i*(curWidth+1)] = original[(curWidth-1) + i*curWidth];
            rightChild[(curWidth) + i*(curWidth+1)] = original[(curWidth-1) + i*curWidth];
          }
          if(threadIdx.x == 0)
          {
            leftChild[leavingVariable+(curWidth+1)*curHeight] = 1.;
            leftChild[curWidth-1+(curWidth+1)*curHeight] = 1.;
            leftChild[(curWidth+1)*(curHeight+1)-1] = floor(breakpoint);
            rightChild[leavingVariable+(curWidth+1)*curHeight] = 1.;
            rightChild[curWidth-1+(curWidth+1)*curHeight] = -1.;
            rightChild[(curWidth+1)*(curHeight+1)-1] = -1*ceil(breakpoint);
          }
          double swap;
          for(int i = threadIdx.x; i < (curWidth+1); i+= blockDim.x)
          {
            swap = leftChild[(curWidth+1) * (curHeight+1) - 2*(curWidth+1) + i];
            leftChild[(curWidth+1) * (curHeight+1) - 2*(curWidth+1) + i] = leftChild[(curWidth+1) * (curHeight+1) - (curWidth+1) + i];
            leftChild[(curWidth+1) * (curHeight+1) - (curWidth+1) + i] = swap;

            swap = rightChild[(curWidth+1) * (curHeight+1) - 2*(curWidth+1) + i];
            rightChild[(curWidth+1) * (curHeight+1) - 2*(curWidth+1) + i] = rightChild[(curWidth+1) * (curHeight+1) - (curWidth+1) + i];
            rightChild[(curWidth+1) * (curHeight+1) - (curWidth+1) + i] = swap;
          }
          // for(int i = 0; i <28; i++)
          // {
          //   printf("%f | ",leftChild[i]);
          //   if(i%7 == 6){printf("\n");}
          // }
          // printf("\n");
          // for(int i = 0; i <28; i++)
          // {
          //   printf("%f | ",rightChild[i]);
          //   if(i%7 == 6){printf("\n");}
          // }
          atomicAdd(otherLengthCPU,2);
          for(int i = threadIdx.x; i < (curWidth+1)*(curHeight+1);i+=blockDim.x)
          {
            othertableList[(*otherLengthCPU - 2)*(curWidth+1)*(curHeight+1) + i] = leftChild[i];
            othertableList[(*otherLengthCPU - 1)*(curWidth+1)*(curHeight+1) + i] = rightChild[i];
          }
        }

      }//after for loop, I will run out of job, I will wait for other to be done
      if (threadIdx.x == 0){atomicAdd(doneCPU, 1);localDone = true;}
      __syncthreads();
    }
    __syncthreads();
  }
  //}
}


int main ()
{
  int numblocks = 1;
  int safeCount = 0;
  int height = 3;
  int width = 6;

  double *tableList1, * tableList2;
  int * done1;
  int* done2;
  int * length1;
  int * length2;


  hipMallocManaged(&tableList1, sizeof(double) * 20 * width * height);
  hipMallocManaged(&tableList2, sizeof(double) * 20 * (width + 1) * (height + 1));
  hipMallocManaged(&done1, sizeof(int));
  hipMallocManaged(&done2, sizeof(int));
  hipMallocManaged(&length1, sizeof(int));
  hipMallocManaged(&length2, sizeof(int));


  tableList1[0] = 10.;
  tableList1[1] = 7.;
  tableList1[2] = 1.;
  tableList1[3] = 0.;
  tableList1[4] = 0.;
  tableList1[5] = 40.;

  tableList1[6] = 1.;
  tableList1[7] = 1.;
  tableList1[8] = 0.;
  tableList1[9] = 1.;
  tableList1[10] = 0.;
  tableList1[11] = 5.;

  tableList1[12] = -17.;
  tableList1[13] = -12.;
  tableList1[14] = 0.;
  tableList1[15] = 0.;
  tableList1[16] = 1.;
  tableList1[17] = 0.;
  *length1 = 1;
  *length2 = 0;
  *done1 = 0;
  *done2 = numblocks;
  //CPU Logic
  int *done;
  double *tableList;
  double *othertableList;
  int *length;
  int *otherLength;

  hipMallocManaged(&done, sizeof(int));
  hipMallocManaged(&length, sizeof(int));
  hipMallocManaged(&otherLength, sizeof(int));

  while(!(((*done2 == numblocks) && (*length1 == 0)) || ((*done1 == numblocks) && (*length2 == 0)) || safeCount >= 10))
  {
    hipMallocManaged(&tableList, sizeof(double) * 20 * width * height);
    hipMallocManaged(&othertableList, sizeof(double) * 20 * (width + 1) * (height + 1));
    safeCount++;
    if (*done1 == numblocks){
      done = done2;
      tableList = tableList2;
      othertableList = tableList1;
      length = length2;
      otherLength = length1;
    }
    else{
      done = done1;
      tableList = tableList1;
      othertableList = tableList2;
      length = length1;
      otherLength = length2;
    }
    *otherLength = 0;
    *done1 = 0;
    *done2 = 0;

    // printf("Kernel called\n");
    branchAndBound<<<numblocks,1>>> (numblocks, done, length, otherLength, tableList, othertableList,
      width,height);
    hipDeviceSynchronize();
    width++;
    height++;
    hipFree(tableList);
    hipFree(othertableList);
  }
  // for(int i = 0; i <28; i++)
  // {
  //   printf("%f | ",(othertableList[i]));
  //   if(i%7 == 6){printf("\n");}
  // }
  // printf("\n");
  // for(int i = 0; i <28; i++)
  // {
  //   printf("%f | ",othertableList[28+i]);
  //   if(i%7 == 6){printf("\n");}
  // }
  // cout <<endl;
  // for(int i = 0; i <18; i++)
  // {
  //   printf("%f | ",tableList1[i]);
  //   if(i%6 == 5){printf("\n");}
  // }

  // branchAndBound<<<numblocks, 1>>> (numblocks,10, done1, done2, length1, length2, tableList1, tableList2);
  // hipDeviceSynchronize();
  // for (int i = 0; i < 10; i++)
  // {
  //   cout << tableList1[i] << " ";
  // }
  //   cout << endl;
  //
  // for (int i = 0; i < 10; i++)
  //   cout << tableList2[i] << " ";
  //   cout << endl;

  hipFree(done1);
  hipFree(done2);
  hipFree(length1);
  hipFree(length2);
  hipFree(tableList1);
  hipFree(tableList2);
  hipFree(done);
  hipFree(length);
  hipFree(otherLength);
  return 0;
}
