
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

using namespace std;

__global__ void smallSimplex(double* job, int width, int height)
{
  extern __shared__ double shared[];
  double* pivotColumn = &shared[0];
  double* pivotRow = &shared[height];
  double* ratioColumn = &shared[width];


  __shared__ double smallestObj;
  __shared__ int pivotColIndx;
  __shared__ int pivotRowIndx;

  smallestObj = 10^20;
  int index = blockDim.x * blockIdx.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  while(true)
  {
    //  STEP 1
    if(threadIdx.x == 0)
    {
      for(int i = 0; i < width; i++)
      {
        if(smallestObj > job[(height-1)*width+i])
        {
          smallestObj = job[(height-1)*width+i];
          pivotColIndx = i;
        }
      }
    }
    __syncthreads();
    if(smallestObj >= 0){break;}
    // printf("Smallest collumn is %d with value %f \n",pivotColIndx,smallestObj);

    for(int i = index; i < height; i+=stride)
    {
        pivotColumn[i] = job[pivotColIndx + i*width];
        ratioColumn[i] = job[(i+1)*width-1] / pivotColumn[i];
    }

    // printf("The ratio column is:\n");
    // for(int i = 0; i < height; i++)
    // {
    //   printf("%f\n",ratioColumn[i]);
    // }
  //  STEP 2
    smallestObj = 10^20;
    if(threadIdx.x == 0)
    {
      for(int i = 0; i < height-1; i++)
      {
        if(smallestObj > ratioColumn[i])
        {
          smallestObj = ratioColumn[i];
          pivotRowIndx = i;
        }
      }
    }
    __syncthreads();
    // printf("Smallest row is %d with value %f \n",pivotRowIndx,smallestObj);
    for(int i = index; i < width; i+=stride)
    {
      job[width*pivotRowIndx + i] = job[width*pivotRowIndx + i]/pivotColumn[pivotRowIndx];
      pivotRow[i] = job[width*pivotRowIndx + i];
    }
  //  STEP 3
    for (int i = index; i < height; i+= stride)
    {
      if(i == pivotRowIndx){continue;}
      for (int j = 0; j < width; j++)
      {
        job[width * i + j] = job[width * i + j] - pivotColumn[i]* pivotRow[j];
      }
    }
  }

}

int main(int argc, char const *argv[])
{
  double* simplexTable;
  hipMallocManaged(&simplexTable, sizeof(double)*18);
  // simplexTable[0] = 3.;
  // simplexTable[1] = 4.;
  // simplexTable[2] = 1.;
  // simplexTable[3] = 0.;
  // simplexTable[4] = 0.;
  // simplexTable[5] = 24.;
  //
  // simplexTable[6] = 7.;
  // simplexTable[7] = -4.;
  // simplexTable[8] = 0.;
  // simplexTable[9] = 1.;
  // simplexTable[10] = 0.;
  // simplexTable[11] = 16.;
  //
  // simplexTable[12] = -2.;
  // simplexTable[13] = 3.;
  // simplexTable[14] = 0.;
  // simplexTable[15] = 0.;
  // simplexTable[16] = 1.;
  // simplexTable[17] = 0.;

  simplexTable[0] = 10.;
  simplexTable[1] = 7.;
  simplexTable[2] = 1.;
  simplexTable[3] = 0.;
  simplexTable[4] = 0.;
  simplexTable[5] = 40.;

  simplexTable[6] = 1.;
  simplexTable[7] = 1.;
  simplexTable[8] = 0.;
  simplexTable[9] = 1.;
  simplexTable[10] = 0.;
  simplexTable[11] = 5.;

  simplexTable[12] = -17.;
  simplexTable[13] = -12.;
  simplexTable[14] = 0.;
  simplexTable[15] = 0.;
  simplexTable[16] = 1.;
  simplexTable[17] = 0.;
  int height = 3;
  int width = 6;
  int sharedMemory = sizeof(double)*(height*2 + width);
  smallSimplex<<<1,256,sharedMemory>>>(simplexTable,width,height);
  hipDeviceSynchronize();
  for(int i = 0; i <18; i++)
  {
    cout<<simplexTable[i]<<" "<<"|"<<" ";
    if(i%6 == 5){cout<<endl;}
  }
  return 0;
}
